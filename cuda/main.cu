#include "hip/hip_runtime.h"
#include "./cliques.cuh"
#include <iostream>
#include <vector>
#include <map>
#include <fstream>
#include <algorithm>

double read_timer() {
  static int initialized = 0;
  static struct timeval start;
  struct timeval end;
  if (!initialized) {
    gettimeofday(&start, NULL);
    initialized = 1;
  }
  gettimeofday(&end, NULL);
  return (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
}

double start_time, end_time; /* start and end times */

using namespace std;

__global__ void count_cliques(int *fila, int *fila_index, int *count, int *flat_adj_list_arr, int *offsets_arr, int *cliques){
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  int filaIndex = atomicAdd(fila_index, 1);
  int startVertex = fila[filaIndex];

  int clique[MAX_CLIQUE_SIZE];
  clique[0] = startVertex;

  int startIndex = thread_id * MAX_CLIQUES * MAX_CLIQUE_SIZE;

  cliques[startIndex] = clique[0];
  int qntCliques = 1;
  int inicioCliques = 0;

  bool filled = false;

  while(qntCliques - inicioCliques >0){
    // Indice da primeira clique das cliques dessa thread
    int cliqueIndex = startIndex + inicioCliques * MAX_CLIQUE_SIZE;
    // Move o inicio das cliques para a próxima clique
    inicioCliques++;

    // Calcula o tamanho da clique atual
    int cliqueSize = 0;
    for(int i = 0; i < MAX_CLIQUE_SIZE; i++){
      if(cliques[cliqueIndex + i] != -1){
        cliqueSize++;
      }
    }

    // Se a clique atual já tem o tamanho máximo, incrementa o contador e passa para a próxima clique
    if(cliqueSize == MAX_CLIQUE_SIZE){
      atomicAdd(count, 1);
      continue;
    }
    
    // Pega o último vértice da clique atual
    int lastVertex = cliques[cliqueIndex + cliqueSize - 1];

    // Percorre os vértices da clique atual
    for(int i = 0; i < cliqueSize; i++){
      // Pega o vértice atual
      int vertexAtual = cliques[cliqueIndex + i];

      // Percorre os vizinhos do vértice atual
      for(int j = offsets_arr[vertexAtual]; j < offsets_arr[vertexAtual + 1]; j++){
        // Pega o vizinho
        int vizinho = flat_adj_list_arr[j];

        // Se o vizinho for maior que o último vértice da clique
        if(vizinho > lastVertex){

          // Verifica se o vizinho já está na clique
          bool isInClique = false;


          // Percorre os vértices da clique
          for(int k = 0; k < cliqueSize; k++){
            // Se o vizinho já está na clique
            if(cliques[cliqueIndex + k] == vizinho){
              isInClique = true;
              break;
            }
          }

          // Se o vizinho não está na clique
          if(!isInClique){

            // Verifica se o vizinho é vizinho de todos os vértices da clique
            bool ehVizinhoDeTodos = true;

            // Percorre os vértices da clique
            for(int k = 0; k < cliqueSize; k++){
              // Pega o vértice da clique
              int vertexClique = cliques[cliqueIndex + k];

              bool ehVizinho = false;

              // Percorre os vizinhos do vértice da clique
              for(int l = offsets_arr[vertexClique]; l < offsets_arr[vertexClique + 1]; l++){
                // Se o vizinho do vértice da clique for o vizinho
                if(flat_adj_list_arr[l] == vizinho){
                  ehVizinho = true;
                  break;
                }
              }

              // Se o vizinho não for vizinho de um dos vértices da clique, não é vizinho de todos
              if(!ehVizinho){
                ehVizinhoDeTodos = false;
                break;
              }
            }

            // Se o vizinho for vizinho de todos os vértices da clique
            if(ehVizinhoDeTodos){

              // Cria uma nova clique com o vizinho
              int newClique[MAX_CLIQUE_SIZE];
              for(int k = 0; k < cliqueSize; k++){
                newClique[k] = cliques[cliqueIndex + k];
              }
              newClique[cliqueSize] = vizinho;

              // add -1 to the end of the clique if it needs, considering MAX_CLIQUE_SIZE can change
              for(int k = cliqueSize + 1; k < MAX_CLIQUE_SIZE; k++){
                newClique[k] = -1;
              }

              // Verifica se a nova clique já existe
              bool cliqueJaExiste = false;

              // Percorre as cliques
              for(int k = 0; k < qntCliques; k++){
                // Pega a clique atual
                int cliqueAtual[MAX_CLIQUE_SIZE];
                for(int l = 0; l < MAX_CLIQUE_SIZE; l++){
                  cliqueAtual[l] = cliques[startIndex + k * MAX_CLIQUE_SIZE + l];
                }

                // Verifica se a nova clique é igual a clique atual
                bool saoIguais = true;
                for(int l = 0; l < MAX_CLIQUE_SIZE; l++){
                  if(newClique[l] != cliqueAtual[l]){
                    saoIguais = false;
                    break;
                  }
                }

                // Se a nova clique é igual a clique atual, a nova clique já existe
                if(saoIguais){
                  cliqueJaExiste = true;
                  break;
                }
              }

              if(!cliqueJaExiste){
                // Adiciona a nova clique
                if(startIndex + qntCliques * MAX_CLIQUE_SIZE + MAX_CLIQUE_SIZE >= (startIndex + MAX_CLIQUES * MAX_CLIQUE_SIZE)){
                  printf("ERRO: Thread %d, encheu o seu espaço de cliques, parando para evitar de usar memória de outra thread. Ou seja, desconsiderando as cliques iniciadas pelo vértice %d\n", thread_id, startVertex);
                  filled = true;
                  break;
                }
                for(int k = 0; k < MAX_CLIQUE_SIZE; k++){
                  cliques[startIndex + qntCliques * MAX_CLIQUE_SIZE + k] = newClique[k];
                }
              }

              qntCliques++;
            }
          }
        }

        if(filled){
          break;
        }
      }

      if(filled){
        break;
      }
    }

    if(filled){
      break;
    }
  }
}

int main(int argc, char *argv[]){

  if (argc < 2){
    cout << "Usage: " << argv[0] << " <input_file>" << endl;
    return 1;
  }

  cout << "Usando valor de MAX_CLIQUE_SIZE = " << MAX_CLIQUE_SIZE << endl;
  cout << "Usando valor de MAX_CLIQUES = " << MAX_CLIQUES << endl;
  cout << "Caso deseje alterar, altere os valores das constantes MAX_CLIQUE_SIZE e MAX_CLIQUES no arquivo cliques.cuh" << endl;

  string filename = argv[1];

  int num_vertices = 0;
  vector<int> vertices;
  vector<vector<int>> adj_list;

  readGraph(filename, adj_list, num_vertices);

  vector<int> flat_adj_list;
  vector<int> offsets;

  flatten(adj_list, flat_adj_list, offsets);

  int *flat_adj_list_arr = (int *)malloc(flat_adj_list.size() * sizeof(int));
  toArray(flat_adj_list, flat_adj_list_arr);

  int *offsets_arr = (int *)malloc(offsets.size() * sizeof(int));
  toArray(offsets, offsets_arr);

  // print adj list from array
  // for(int i = 0; i < num_vertices; i++){
  //   cout << i << ": ";
  //   for(int j = offsets_arr[i]; j < offsets_arr[i+1]; j++){
  //     cout << flat_adj_list_arr[j] << " ";
  //   }
  //   cout << endl;
  // }
  //

  int *fila_h = (int *)malloc(num_vertices * sizeof(int));

  for(int i = 0; i < num_vertices; i++){
    fila_h[i] = i;
  }

    // Alloc and copy fila_h to device
  int *fila_d;
  hipMalloc((void **)&fila_d, num_vertices * sizeof(int));
  hipMemcpy(fila_d, fila_h, num_vertices * sizeof(int), hipMemcpyHostToDevice);

  int fila_index_h = 0;
  int *fila_index_d;
  hipMalloc((void **)&fila_index_d, sizeof(int));
  hipMemcpy(fila_index_d, &fila_index_h, sizeof(int), hipMemcpyHostToDevice);

  int count_h = 0;
  int *count_d;
  hipMalloc((void **)&count_d, sizeof(int));
  hipMemcpy(count_d, &count_h, sizeof(int), hipMemcpyHostToDevice);

  int *flat_adj_list_arr_d;
  hipMalloc((void **)&flat_adj_list_arr_d, flat_adj_list.size() * sizeof(int));
  hipMemcpy(flat_adj_list_arr_d, flat_adj_list_arr, flat_adj_list.size() * sizeof(int), hipMemcpyHostToDevice);

  int *offsets_arr_d;
  hipMalloc((void **)&offsets_arr_d, offsets.size() * sizeof(int));
  hipMemcpy(offsets_arr_d, offsets_arr, offsets.size() * sizeof(int), hipMemcpyHostToDevice);

  int block_size = 64;
  int amount_of_blocks = num_vertices / block_size;

  int num_threads = block_size * amount_of_blocks; // basicamente o número de vértices, ta em outra variável só pq o código é meu

  int *cliques_d;
  // Aloca memória para as cliques
  hipMalloc((void **)&cliques_d, num_threads * MAX_CLIQUES * MAX_CLIQUE_SIZE * sizeof(int));
  hipMemset(cliques_d, -1, num_threads * MAX_CLIQUES * MAX_CLIQUE_SIZE * sizeof(int));

  count_cliques<<<amount_of_blocks, block_size>>>(fila_d, fila_index_d, count_d, flat_adj_list_arr_d, offsets_arr_d, cliques_d);
  start_time = read_timer();
  hipDeviceSynchronize();

  end_time = read_timer();
  hipMemcpy(&count_h, count_d, sizeof(int), hipMemcpyDeviceToHost);

  int *cliques_h = (int *)malloc(num_threads * MAX_CLIQUES * MAX_CLIQUE_SIZE * sizeof(int));
  hipMemcpy(cliques_h, cliques_d, num_threads * MAX_CLIQUES * MAX_CLIQUE_SIZE * sizeof(int), hipMemcpyDeviceToHost);

  cout << "Number of cliques: " << count_h << endl;
  cout << "Time: " << end_time - start_time << " seconds" << endl;

  free(flat_adj_list_arr);
  free(offsets_arr);
  free(fila_h);
  free(cliques_h);
  
  hipFree(fila_d);
  hipFree(fila_index_d);
  hipFree(count_d);
  hipFree(flat_adj_list_arr_d);
  hipFree(offsets_arr_d);
  hipFree(cliques_d);

  return 0;

}
